#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

/* GPU kernel for in-place GEMM operation */
__global__
void myGEMM_kernel(double* A, double* B, double* C,
                   double alpha, double beta,
                   int M, int N, int K,
                   bool AT, bool BT) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int c_ind = row + (col * M);
        double dot_prod = 0.0;
        int a_ind;
        int b_ind;
        for(int i = 0; i < K; i++) {
            if (AT)
                a_ind = (row*K) + i;
            else
                a_ind = row + (i*M);
            if (BT)
                b_ind = col + (i*N);
            else
                b_ind = i + (col * K);
            dot_prod += A[a_ind] * B[b_ind];
        }
        C[c_ind] = (alpha * dot_prod) + (beta * C[c_ind]);
    }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* A, double* B, double* C,
           double* alpha, double* beta,
           int M, int N, int K,
           bool AT, bool BT) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    myGEMM_kernel<<< blocks, threads >>>(A, B, C, *alpha, *beta, M, N, K, AT, BT);
    check_launch("myGEMM_kernel");
    return 0;
}

/* GPU kernel for 10-class softmax */
__global__
void gpuSoftmax_kernel(double* A, unsigned int num_classes, unsigned int N) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (col < N) {
        double denominator = 0.0;

        for(int c = 0; c < num_classes; c++){
            denominator += (double) std::exp(A[col*num_classes + c]);
        }

        for(int c = 0; c < num_classes; c++){
            int ij = c + (col * num_classes);
            A[ij] = (double) std::exp(A[ij])/ (double) denominator;
        }
    }
}

/* Routine for 10-class softmax */
void gpuSoftmax(double* A, unsigned int num_classes, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = num_threads;
    dim3 threads(thr_x);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    dim3 blocks(blk_x);

    gpuSoftmax_kernel<<< blocks, threads >>>(A, num_classes, N);
    check_launch("gpuSoftmax_kernel");
}

/* GPU kernel for in-place element-wise sigmoid */
__global__
void gpuSigmoid_kernel(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < N && row < num_neurons) {
        int ij = row + (col * num_neurons);
        A[ij] = (double) 1.0 / (double)(1.0 + exp(-1.0 * A[ij]));
    }
}

/* Routine for in-place element-wise sigmoid */
void gpuSigmoid(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (num_neurons + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuSigmoid_kernel<<< blocks, threads >>>(A, num_neurons, N);
    check_launch("gpuSigmoid_kernel");
}

/* GPU kernel for summing rows of matrix A. Places row sums in vector v*/
__global__
void gpuRowSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M) {
        double rowSum = 0.0;
        for (int i = 0; i < N; i++) 
            rowSum += A[(M*i) + row];
        v[row] = rowSum;
    }
}

/* Routine for summing rows of matrix A. Places row sums in vector v */
void gpuRowSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = num_threads;
    dim3 threads(thr_x);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    dim3 blocks(blk_x);

    gpuRowSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuRowSum_kernel");
}

/* GPU kernel for broadcasting sum for matrix A with vector v */
__global__
void gpuMatVecSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; 
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M &&  col < N) {
        int ind = row + (M*col);
        double num = v[row];
        A[ind] += num;
    }
}

/* Routine for broadcasting sum for matrix A with vector v */
void gpuMatVecSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);
    gpuMatVecSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuMatVecSum_kernel");
}

/* GPU kernel for elementwise matrix sum */
__global__
void gpuElementwiseSum_kernel(double *A, double *B, double *C,
                              double alpha, double beta,
                              int M, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < M && col < N) {
        int ind = row + (M*col);
        C[ind] = (alpha * A[ind]) + (beta * B[ind]);
    }
}

/* Routine for elementwise matrix sum */
void gpuElementwiseSum(double *A, double *B, double *C, 
                       double alpha, double beta,
                       int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuElementwiseSum_kernel<<< blocks, threads >>>(A, B, C, alpha, beta, M, N);
    check_launch("gpuElementwiseSum_kernel");
}

/* GPU kernel for derivative of sigmoid */
__global__
void gpudSigmoid_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int ind = row + (M*col);
        C[ind] = (double) A[ind] * B[ind] * (1.0 - B[ind]);
    }
}

/** Routine for derivative of sigmoid */
void gpudSigmoid(double *A, double *B, double *C, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpudSigmoid_kernel<<< blocks, threads >>>(A, B, C, M, N);
    check_launch("gpudSigmoid");
}
