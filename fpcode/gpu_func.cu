#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

/* GPU kernel for in-place GEMM operation */
__global__
void myGEMM_kernel(double* A, double* B, double* C,
                   double alpha, double beta,
                   int M, int N, int K,
                   bool AT, bool BT, bool CZ) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (!(row > M || col > N)) {
        int c_ind = row + (col * M);
        double dot_prod = 0.0;
        int a_ind;
        int b_ind;
        for(int i = 0; i < K; i++) {
            if (AT)
                a_ind = (row*K) + i;
            else
                a_ind = row + (i*M);
            if (BT)
                b_ind = col + (i*N);
            else
                b_ind = i + (col * K);
            dot_prod += A[a_ind] * B[b_ind];
        }
        if (CZ)
            C[c_ind] = (alpha * dot_prod);
        else
            C[c_ind] = (alpha * dot_prod) + (beta * C[c_ind]);
    }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* A, double* B, double* C,
           double* alpha, double* beta,
           int M, int N, int K,
           bool AT, bool BT, bool CZ) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    myGEMM_kernel<<< blocks, threads >>>(A, B, C, *alpha, *beta, M, N, K, AT, BT, CZ);
    check_launch("myGEMM_kernel");
    return 0;
}

/* GPU kernel for 10-class softmax */
__global__
void gpuSoftmax_kernel(double* A, unsigned int num_classes, unsigned int N) {
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (!(col > N)) {
        double denominator = 0.0;

        for(unsigned int c = 0; c < num_classes; c++){
            unsigned int ij = c + (col * num_classes);
            denominator += (double) std::exp(A[ij]);
        }

        for(unsigned int c = 0; c < num_classes; c++){
            unsigned int ij = c + (col * num_classes);
            A[ij] = (double) std::exp(A[ij]) / denominator;
        }
    }
}

/* Routine for 10-class softmax */
void gpuSoftmax(double* A, unsigned int num_classes, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (num_classes + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuSoftmax_kernel<<< blocks, threads >>>(A, num_classes, N);
    check_launch("gpuSoftmax_kernel");
}

/* GPU kernel for in-place element-wise sigmoid */
__global__
void gpuSigmoid_kernel(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(!(col > N || row > num_neurons)) {
        int ij = row + (col * num_neurons);
        A[ij] = (double) 1.0 / (double)(1.0 + exp(-1.0 * A[ij]));
    }
}

/* Routine for in-place element-wise sigmoid */
void gpuSigmoid(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (num_neurons + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuSigmoid_kernel<<< blocks, threads >>>(A, num_neurons, N);
    check_launch("gpuSigmoid_kernel");
}

/* GPU kernel for summing rows of matrix A. Places row sums in vector v*/
__global__
void gpuRowSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (!(row > M)) {
        double rowSum = 0.0;
        int ind;
        for (int i = 0; i < N; i++) {
            ind = row + (M*i);
            rowSum += A[ind];
        }
        v[row] = rowSum;
    }
}

/* Routine for summing rows of matrix A. Places row sums in vector v */
void gpuRowSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = num_threads;
    dim3 threads(thr_x);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    dim3 blocks(blk_x);

    gpuRowSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuRowSum_kernel");
}

/* GPU kernel for broadcasting sum for matrix A with vector v */
__global__
void gpuMatVecSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if (!(row > M || col > N)) {
        int ind = row + (M*col);
        double num = v[row];
        A[ind] += num;
    }
}

/* Routine for broadcasting sum for matrix A with vector v */
void gpuMatVecSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuMatVecSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuMatVecSum_kernel");
}

/* GPU kernel for elementwise Hadamard product */
__global__
void gpuHadamard_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (!(row > M || col > N)) {
        int ind = row + (M*col);
        C[ind] = A[ind] * B[ind];
    }
}

/* Routine for elementwise Hadamard product */
void gpuHadamard(double *A, double *B, double *C, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuHadamard_kernel<<< blocks, threads >>>(A, B, C, M, N);
    check_launch("gpuHadamard_kernel");
}

/* GPU kernel for elementwise matrix sum */
__global__
void gpuElementwiseSum_kernel(double *A, double *B, double *C,
                              double alpha, double beta,
                              int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (!(row > M || col > N)) {
        int ind = row + (M*col);
        C[ind] = (alpha * A[ind]) + (beta * B[ind]);
    }
}

/* Routine for elementwise matrix sum */
void gpuElementwiseSum(double *A, double *B, double *C, 
                       double alpha, double beta,
                       int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuElementwiseSum_kernel<<< blocks, threads >>>(A, B, C, alpha, beta, M, N);
    check_launch("gpuElementwiseSum_kernel");
}

/* GPU kernel for in-place matrix scalar prodcut */
__global__
void gpuMatrixScalarProduct_kernel(double *A, double alpha, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (!(row > M || col > N)) {
        int ind = row + (M*col);
        A[ind] = (alpha * A[ind]);
    }
}

/* Routine for in-place matrix scalar product */
void gpuMatrixScalarProduct(double *A, double alpha, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuMatrixScalarProduct_kernel<<< blocks, threads >>>(A, alpha, M, N);
    check_launch("gpuMatrixScalarProduct_kernel");
}

/* GPU kernel for derivative of sigmoid */
__global__
void gpudSigmoid_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (!(row > M || col > N)) {
        int ind = row + (M*col);
        C[ind] = (double) A[ind] * B[ind] * (1.0 - B[ind]);
    }
}

/** Routine for derivative of sigmoid */
void gpudSigmoid(double *A, double *B, double *C, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 16;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpudSigmoid_kernel<<< blocks, threads >>>(A, B, C, M, N);
    check_launch("gpudSigmoid");
}
